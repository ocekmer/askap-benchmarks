#include "hip/hip_runtime.h"
#include "WarmupGPU.h"

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void warmup(int N)
{
    //int N = 1<<8;

    float *x, *y, *d_x, *d_y;
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}
