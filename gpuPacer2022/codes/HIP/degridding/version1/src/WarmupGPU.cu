#include "hip/hip_runtime.h"
#include "WarmupGPU.h"

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

__global__
void vector_add(float *out, float *a, float *b, int n)
{
    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

void warmup(int N)
{
    N = 1<<N;

    float *x, *y, *d_x, *d_y, *out, *d_out;
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));
    out = (float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));
    hipMalloc(&d_out, N*sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    //saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
    vector_add<<<1,1>>>(d_out, d_x, d_y, N);

    //hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(out, d_out, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_out);
    free(x);
    free(y);
    free(out);
}
